#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void kernel(int a, int b, int *c)
{
	*c = a + b;
}

int main(void)
{
	int c;
	int *dev_c;
	hipMalloc((void**)&dev_c, sizeof(int));
	kernel << <1, 1 >> > (2, 7, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2+7=%d\n",c);
	hipFree(dev_c);

	int res;
	int *dev_res;
	hipMalloc((void**)&dev_res, sizeof(int));
	kernel << <1, 1 >> > (8, 2, dev_res);
	hipMemcpy(&res, dev_res, sizeof(int), hipMemcpyDeviceToHost);
	cout << *dev_res << endl;
	cout << "8+2=" << res << endl;

	return 0;
    
}
