#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

int main()
{
	hipDeviceProp_t prop;

	int count;
	hipGetDeviceCount(&count);
	for (int i = 0; i < count; i++)
	{
		hipGetDeviceProperties(&prop, i);

		cout << "information for device " << i << endl;
		cout << "name:" << prop.name << endl;
		cout << "capability:" << prop.major << "." << prop.minor << endl;
		cout << "clock rate:" << prop.clockRate << endl;
		hipDeviceProp_t sDevProp = prop;
		printf("%d \n", i);
		printf("Device name: %s\n", sDevProp.name);
		printf("Device memory: %d\n", sDevProp.totalGlobalMem);
		printf("Memory per-block: %d\n", sDevProp.sharedMemPerBlock);
		printf("Register per-block: %d\n", sDevProp.regsPerBlock);
		printf("Warp size: %d\n", sDevProp.warpSize);
		printf("Memory pitch: %d\n", sDevProp.memPitch);
		printf("Constant Memory: %d\n", sDevProp.totalConstMem);
		printf("Max thread per-block: %d\n", sDevProp.maxThreadsPerBlock);
		printf("Max thread dim: ( %d, %d, %d )\n", sDevProp.maxThreadsDim[0],
			sDevProp.maxThreadsDim[1], sDevProp.maxThreadsDim[2]);
		printf("Max grid size: ( %d, %d, %d )\n", sDevProp.maxGridSize[0],
			sDevProp.maxGridSize[1], sDevProp.maxGridSize[2]);
		printf("Ver: %d.%d\n", sDevProp.major, sDevProp.minor);
		printf("Clock: %d\n", sDevProp.clockRate);
		printf("textureAlignment: %d\n", sDevProp.textureAlignment);
	}

	cout << "next" << endl;
	int dev;
	hipGetDevice(&dev);
	cout << "id of current cuda device:" << dev << endl;
	hipDeviceProp_t temp_prop;
	memset(&temp_prop, 0, sizeof(hipDeviceProp_t));
	temp_prop.major = 1;
	temp_prop.minor = 3;
	hipChooseDevice(&dev, &prop);
	cout << "id of cuda device closest to revision 1.3:" << dev << endl;
	hipSetDevice(dev);
}
