
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

using namespace std;

struct MyStruct
{
	int a;
	int b;
	int c;
};

int main(void)
{
	int *ptr;
	char *ptr1;
	int **ptr2;
	int(*ptr3)[3];
	int *(*ptr4)[4];

	int temp = sizeof(*ptr);
	printf("result is :%d\n",temp);

	int array[20];
	int *ptr5 = array;

	for (int i = 0; i < 20; i++)
	{
		array[i] = i;
	}

	for (int i = 0; i < 20; i++)
	{
		(*ptr5)++;
		ptr5++;
	}

	for (int i = 0; i < 20; i++)
	{
		printf("number %d: value %d\n", i, array[i]);
	}

	int a1 = 12;
	int b1;
	int *p1;
	int **ptr6;
	p1 = &a1;
	*p1 = 24;
	ptr6 = &p1;
	*ptr6 = &b1;
	**ptr6 = 34;

	printf("a1 is %d,b1 is %d,*p1 is %d,**ptr6 is %d\n", a1, b1, *p1, **ptr6);

	char *str1[3] = {
		"Hello,this is a sample!\n",
		"Hi,good morning\n",
		"Hello world\n"
	};
	char s[80];
	strcpy(s, str1[0]);
	printf(s);
	cout << **str1 << endl;
	strcpy(s, str1[1]);
	printf(s);
	strcpy(s, str1[2]);
	printf(s);

	MyStruct ss = { 20,30,40 };
	MyStruct *ptr7 = &ss;
	int *pstr = (int*)&ss;
	cout << sizeof(ptr7) << " and " << sizeof(pstr) << endl;
	cout << ptr7->a << " " << ptr7->b << " " << ptr7->c << endl;
	cout << *pstr << " " << *(pstr + 1) << " " << *(pstr + 2) << endl;

	int fun1(char*, int);
	int(*pfun1)(char*, int);
	pfun1 = fun1;
	int res = (*pfun1)("abcdefg", 7);
}

int fun1(char* x1, int x2){
	cout << x1 << " " << x2 << endl;
	return 0;
}
