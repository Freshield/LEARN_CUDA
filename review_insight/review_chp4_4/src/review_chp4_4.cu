/*
 ============================================================================
 Name        : review_chp4_4.cu
 Author      : freshield
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include "../../common/book.h"
#include "../../common/cpu_bitmap.h"

#define DIM 1000

int main(){
	CPUBitmap bitmap(DIM, DIM);
	unsigned char *dev_bitmap;
	
	hipMalloc((void**)&dev_bitmap, bitmap.image_size());

	dim3 grid(DIM, DIM);
	kernel<<<grid, 1>>>(dev_bitmap);

	hipMemcpy(bitmap.get_ptr(), dev_bitmap,
			bitmap.image_size(), hipMemcpyDeviceToHost);

	bitmap.display_and_exit();

	hipFree(dev_bitmap);


}
