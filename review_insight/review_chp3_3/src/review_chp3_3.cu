/*
 ============================================================================
 Name        : review_chp3_3.cu
 Author      : freshield
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include <stdio.h>

int main(){
	int count;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&count);

	printf("cuda device number is %d\n", count);

	hipGetDeviceProperties(&prop, 0);

	printf( "   --- General Information for device %d ---\n", 0 );
	        printf( "Name:  %s\n", prop.name );
	        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
	        printf( "Clock rate:  %d\n", prop.clockRate );
	        printf( "Device copy overlap:  " );
	        if (prop.deviceOverlap)
	            printf( "Enabled\n" );
	        else
	            printf( "Disabled\n");
	        printf( "Kernel execution timeout :  " );
	        if (prop.kernelExecTimeoutEnabled)
	            printf( "Enabled\n" );
	        else
	            printf( "Disabled\n" );

	        printf( "   --- Memory Information for device %d ---\n", 0);
	        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
	        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
	        printf( "Max mem pitch:  %ld\n", prop.memPitch );
	        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

	        printf( "   --- MP Information for device %d ---\n", 0 );
	        printf( "Multiprocessor count:  %d\n",
	                    prop.multiProcessorCount );
	        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
	        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
	        printf( "Threads in warp:  %d\n", prop.warpSize );
	        printf( "Max threads per block:  %d\n",
	                    prop.maxThreadsPerBlock );
	        printf( "Max thread dimensions:  (%d, %d, %d)\n",
	                    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	                    prop.maxThreadsDim[2] );
	        printf( "Max grid dimensions:  (%d, %d, %d)\n",
	                    prop.maxGridSize[0], prop.maxGridSize[1],
	                    prop.maxGridSize[2] );
	        printf( "\n" );
}
