/*
 ============================================================================
 Name        : review_chp3_2_bhd.cu
 Author      : freshield
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c){
	*c = a + b;
}

int main(){
	int c;
	int *dev_c;

	hipMalloc((void**)&dev_c, sizeof(int));

	add<<<1,1>>>(2, 7, dev_c);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);

	printf("2 + 7 = %d", c);


	
}
