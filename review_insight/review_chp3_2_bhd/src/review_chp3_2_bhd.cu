/*
 ============================================================================
 Name        : review_chp3_2_bhd.cu
 Author      : freshield
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c){
	*c = a + b;
}

int find(int *a){
	return *a;
}

int main(){
	int c;
	int *dev_c;

	int b = 1;
	int * b_ad;
	int find_b;
	b_ad = &b;
	find_b = find(&b);
	printf("b is %d\n",b);
	printf("b address is %d\n", &b);
	//printf("*b is %d\n", *b); error
	printf("b_ad is %d\n", b_ad);
	printf("&b_ad is %d\n", &b_ad);
	printf("*b_ad is %d\n", *b_ad);
	printf("find_b is %d\n", find_b);
	printf("find_b address is %d\n", &find_b);



	hipMalloc((void**)&dev_c, sizeof(int));

	add<<<1,1>>>(2, 7, dev_c);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(dev_c);

	printf("2 + 7 = %d", c);


	
}
