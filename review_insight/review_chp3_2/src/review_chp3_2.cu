/*
 ============================================================================
 Name        : review_chp3_2.cu
 Author      : freshield
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c){
	*c = a + b;
}

int main(void){
	int c;
	int *dev_c;//define dev_c as a pointer
	hipMalloc( (void**)&dev_c, sizeof(int));//get dev_c address
	//change dev_c address to void** type
	//allocate a address on GPU, and give the address number to dev_c
	//
	//all in all: dev_c is a pointer, get the dev_c address and change
	//to void** type

	add<<<1,1>>>(2, 7, dev_c);//use function and back value to
	//the place where dev_c point

	hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	//get c address, allocate the value which dev_c store and copy to c

	printf( "2 + 7 = %d\n", c);
	hipFree(dev_c);

	return 0;
}
