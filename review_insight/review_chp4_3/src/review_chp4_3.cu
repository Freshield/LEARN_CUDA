/*
 ============================================================================
 Name        : review_chp4_3.cu
 Author      : freshield
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */
#include <stdio.h>

#include "/media/freshield/BUFFER/LEARN_CUDA/cuda_by_example/common/book.h"
#include "/media/freshield/BUFFER/LEARN_CUDA/cuda_by_example/common/cpu_bitmap.h"

#define DIM 100

void kernel(unsigned char *ptr);
int julia(int x, int y);

struct hipComplex{
	float r;
	float i;
	hipComplex(float a, float b) : r(a), i(b) {}
	float magnitude2(void) {return r * r + i * i;}
	hipComplex operator*(const hipComplex& a){
		return hipComplex(r * a.r - i * a.i, i * a.r + r * a.i);
	}
	hipComplex operator+(const hipComplex& a){
		return hipComplex(r + a.r, i + a.i);
	}
};


int main(){
	printf("begin");

	CPUBitmap bitmap(DIM, DIM);

	unsigned char *ptr = bitmap.get_ptr();

	kernel(ptr);

	bitmap.display_and_exit();

}

void kernel( unsigned char *ptr){
	for(int y = 0; y < DIM; y++){
		for(int x = 0; x < DIM; x++){
			int offset = x + y * DIM;

			int juliaValue = julia(x, y);
			ptr[offset * 4 + 0] = 255 * juliaValue;
			ptr[offset * 4 + 1] = 0;
			ptr[offset * 4 + 2] = 0;
			ptr[offset * 4 + 3] = 255;
		}
	}
}

int julia(int x, int y){
	const float scale = 1.5;
	float jx = scale * (float)(DIM / 2 - x) / (DIM / 2);
	float jy = scale * (float)(DIM / 2 - y) / (DIM / 2);

	hipComplex c(-0.8, 0.156);
	hipComplex a(jx, jy);

	int i = 0;
	for(i = 0; i < 200; i++){
		a = a * a + c;
		if(a.magnitude2() > 1000){
			return 0;
		}
	}
	return 1;
}
