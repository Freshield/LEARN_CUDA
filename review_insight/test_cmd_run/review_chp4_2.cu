/*
 ============================================================================
 Name        : review_chp4_2.cu
 Author      : freshield
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100

__global__ void add(int *a, int *b, int *c){
	int tid = blockIdx.x;
	if(tid < N){
		c[tid] = a[tid] + b[tid];
	}
}

__global__ void set_a(int *a){
	int tid = blockIdx.x;
	if(tid < N){
		a[tid] = -tid;
	}
}

__global__ void set_b(int *b){
	int tid = blockIdx.x;
	if(tid < N){
		b[tid] = tid * tid;
	}
}

int main(){
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;
	
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));
/*
	for(int i = 0; i < N; i++){
		a[i] = -i;
		b[i] = i * i;
	}

	cudaMemcpy(dev_a, a, N * sizeof(int), cudaMemcpyHostToDevice);
	cudaMemcpy(dev_b, b, N * sizeof(int), cudaMemcpyHostToDevice);
*/
	set_a<<<N,1>>>(dev_a);

	set_b<<<N,1>>>(dev_b);

	add<<<N,1>>>(dev_a, dev_b, dev_c);


	hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b, dev_b, N * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++){
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
