/*
 ============================================================================
 Name        : review_chp4_1.cu
 Author      : freshield
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 100

void add(int *a, int *b, int *c){
	int tid = 0;
	while(tid < N){
		c[tid] = a[tid] + b[tid];
		tid += 1;
	}
}

int main(void){
	int a[N], b[N], c[N];

	for(int i = 0; i < N; i++){
		a[i] = - i;
		b[i] = i * i;
	}

	add(a, b, c);

	for(int i = 0; i < N; i++){
		printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}
	
	return 0;
}
